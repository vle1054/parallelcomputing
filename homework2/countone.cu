
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void countones(int *in, int *out) {
  __shared__ int temp;
  unsigned int tid = threadIdx.x;
  if (in[tid]==1){
    atomicAdd(&temp,1);
  }
  __syncthreads();
 *out = temp;
}

int main(int argc, char *argv[]){
  int *in, *out; // host copies in and cout
  int *d_in, *d_out; // device copies
FILE *file = fopen(argv[1], "r");
  int data, row, col;
fscanf(file, "%d",&row);
fscanf(file, "%d", &col);

 int size = row * col * sizeof(int);

hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  // Alloc space for host copies of a, b, c and setup input values
  in = (int *)malloc(size);
  out = (int *)malloc(size);

for (int i = 0; i < row*col; i++)  {
    fscanf(file, "%d", &in[i]);
  }

  fclose(file);

// Copy inputs to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  countones <<<1, row*col>>> (d_in, d_out);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

printf("There are %d ones.\n", *out);

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}
