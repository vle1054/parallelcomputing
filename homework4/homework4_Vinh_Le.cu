#include "hip/hip_runtime.h"
/*
Vinh Le
CSCI 440 - Parallel Computing
Homework 4 - CPU GPU SCAN
Colorado School of Mines 2018
*/
#include <iostream>
#include <ctime>
#include <stdlib.h>

using namespace std;

//Set tolerance for the check
#define TOLERANCE 0.001

__global__ void scan (int * arr, int * arr_gpu, int n) {
  extern __shared__ float temp[]; // allocated on invocation
   int thid = threadIdx.x;
  int1 pout = 0, pin = 1;
  // Load input into shared memory.
   // This is exclusive scan, so shift right by one
   // and set first element to 0
  temp[pout*n + thid] = (thid > 0) ? arr[thid-1] : 0;
  __syncthreads();
  for (int offset = 1; offset < n; offset *= 2)
  {
    pout = 1 - pout; // swap double buffer indices
    pin = 1 - pout;
    if (thid >= offset)
      temp[pout*n+thid] += temp[pin*n+thid - offset];
    else
      temp[pout*n+thid] = temp[pin*n+thid];
    __syncthreads();
  }
  arr_gpu[thid] = temp[pout*n+thid]; // write output


}

int main(int argc, char *argv[]){

srand(time(NULL));

int n = atoi(argv[1]);
n = (int) malloc(sizeof(int));
//Generate array
cout<<"Generating "<<n<< " random numbers"<<endl;

int * arr, * arr_cpu, * arr_gpu;
arr = (int *) malloc(n*sizeof(int));
arr_cpu = (int *) malloc(n*sizeof(int));
arr_gpu = (int *) malloc(n*sizeof(int));

//fill arr with rnd nums between 1-1000
for (int i = 0; i<n; i++){
  arr[i]= rand()%1000 + 1;
}

cout<<"CPU SCAN"<<endl;

//set 0th element
arr_cpu[0]=arr[0];

// CPU SCAN
for (int i=1; i<n; i++) {
  arr_cpu[i]= arr_cpu[i-1]+arr[i];
}

cout<<"GPU SCAN"<<endl;

//initialize and allocate memory for device same set as host
int * arr_d, * arr_gpu_d;
int * n_d;

hipMalloc((void**) & arr_d, n*sizeof(int));
hipMalloc((void**) & arr_gpu_d, n*sizeof(int));
hipMalloc((void**) & n_d, sizeof(int));

//copy data from host to device
hipMemcpy(arr_d, arr, n*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(n_d, &n, sizeof(int), hipMemcpyHostToDevice);
//GPU SCAN
scan<<<n, 32>>>(arr_d, arr_gpu_d, n_d);
//copy data from device to host
hipMemcpy(arr_gpu, arr_gpu_d, n*sizeof(float), hipMemcpyDeviceToHost);


//Compares arr_cpu with arr_gpu to determine accuracy
int tfail = 0;
for (int i = 0; i < n; i++) {
  if (abs(arr_gpu[i] - arr_cpu[i]) > TOLERANCE) {//take abs value and compare with tolerance
    tfail += 1;//if difference exceeds tolerance
  }
}

//print the number of failures
cout << "Number of Failures: " << tfail <<"\n";

return 0;
}
