/*
Vinh Le
CSCI 440 - Parallel Computing
Homework 4 - CPU GPU SCAN
Colorado School of Mines 2018
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <stdlib.h>
#include <math.h>
#include <cstdio>

using namespace std;

//Set tolerance for the check
#define TOLERANCE 0.001
#define BLOCK_SIZE 1024

__global__ void scan (int * arr, int * arr_gpu, int n) {
  __shared__ float temp[BLOCK_SIZE];

  int tid = threadIdx.x;
  temp[tid]=arr[tid];

    for (unsigned int stride = BLOCK_SIZE/2; stride > 0;stride /= 2){
      __syncthreads();
      if (tid+stride<BLOCK_SIZE){
        temp[tid+stride] +=temp[tid];
      }
      __syncthreads();
    }
  arr_gpu[tid] = temp[tid];

}

int main(int argc, char *argv[]){

srand(time(NULL));

int n = atoi(argv[1]);

//Generate array
cout<<"Generating "<<n<< " random numbers"<<endl;

int *arr, * arr_cpu, * arr_gpu;
arr = (int *) malloc(n*sizeof(int));
arr_cpu = (int *) malloc(n*sizeof(int));
arr_gpu = (int *) malloc(n*sizeof(int));

//fill arr with rnd nums between 1-1000
for (int i = 0; i<n; i++){
  //arr[i]= rand()%1000 + 1;
arr[i]=1;
}

cout<<"CPU SCAN"<<endl;

//set 0th element
arr[0]=0;
arr_cpu[0]=0;

// CPU SCAN
for (int i=1; i<n; i++) {
  arr_cpu[i]= arr_cpu[i-1]+arr[i];
}

cout<<"GPU SCAN"<<endl;




//initialize and allocate memory for device same set as host
int * arr_d, * arr_gpu_d;


hipMalloc((void**) & arr_d, n*sizeof(int));
hipMalloc((void**) & arr_gpu_d, n*sizeof(int));


//copy data from host to device
hipMemcpy(arr_d, arr, n*sizeof(int), hipMemcpyHostToDevice);
//GPU SCAN
int NUM_BLOCK = ceil(n/BLOCK_SIZE);
printf("%d\n", NUM_BLOCK );
scan<<<NUM_BLOCK, BLOCK_SIZE>>>(arr_d, arr_gpu_d, n);
//copy data from device to host
hipMemcpy(arr_gpu, arr_gpu_d, n*sizeof(float), hipMemcpyDeviceToHost);



//Compares arr_cpu with arr_gpu to determine accuracy
int tfail = 0;
for (int i = 0; i < n; i++) {
  if (abs(arr_gpu[i] - arr_cpu[i]) > TOLERANCE) {//take abs value and compare with tolerance
    tfail += 1;//if difference exceeds tolerance
  }
}

//print the number of failures
cout << "Number of Failures: " << tfail <<"\n";


return 0;
}
