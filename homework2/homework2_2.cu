#include "hip/hip_runtime.h"
/*
Vinh Le
CSCI 440 - Parallel Computing
Colorado School of Mines 2018
*/
#include <stdio.h>

__global__ void transpose(int *in, int *out, int row) {
  __shared__ int *temp;
  unsigned int tid = threadIdx.x;

  out[(tid/row)*row+(tid%row)] = in[tid]

  __syncthreads();
  *out = *temp;
}

int main(int argc, char *argv[]){

  FILE *file = fopen(argv[1], "r");
  int row, col;
  fscanf(file, "%d",&row);
  fscanf(file, "%d", &col);

  int size = row * col * sizeof(int);

  int *in, *out; // host copies in and cout
  in = (int *)malloc(size);
  out = (int *)malloc(size);

  for (int i = 0; i < row*col; i++)  {
    fscanf(file, "%d", &in[i]);
  }

  fclose(file);

  int *d_in, *d_out; // device copies
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  // Copy inputs to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  transpose <<<1, row*col>>> (d_in, d_out, row);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  for (int i = 0; i<col;i++){
    for (int j=0; j< row;j++){
      printf(out[i*col+j])
    }
  }
  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}
