#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <Vector>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void countones(int *in, int *out) {

  __shared__ int *temp;

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

if (in[tid]==1){
  atomicadd(*temp,1);

}

  __syncthreads();

  *out = *temp;
}
int main(int argc, char *argv[]){

  int row, col, temp;
  vector<int> array;

  string infile = argv[1];

  ifstream fin;
  fin.open(infile);
  fin >> row >> col;

  for(ini=0; i<(row*col); i++){
    fin<<temp;
    array[i]=temp;
  }

  cout << "DONE\n";

  fin.close();



  int *a, *b, *c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  countones <<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>> (d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
