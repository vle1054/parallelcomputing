#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void countones(int *in, int *out) {

  __shared__ int *temp;

  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

if (in[tid]==1){
  atomicadd(*temp,1);

}

  __syncthreads();

  *out = *temp;
}

int main(int argc, char *argv[]){

  int *in, *out; // host copies in and cout
  int *d_in, *d_out; // device copies
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  // Alloc space for host copies of a, b, c and setup input values
  in = (int *)malloc(size);
  out = (int *)malloc(size);

  int row, col,temp;
  int *array;

  string infile = argv[1];

  ifstream fin;
  fin.open(infile);
  fin >> row >> col;

  for(ini=0; i<(row*col); i++){
    fin<<temp;
    array[i]=temp;
  }

  cout << "DONE\n";

  fin.close();

  int *in = *array;



  // Copy inputs to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  countones <<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>> (d_in, d_out);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}
