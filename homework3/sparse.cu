#include "hip/hip_runtime.h"
/*
	Vinh Le
	CSCI 440 - Parallel Computing
	Homework 3 - Sparse Vector Matrix
	Colorado School of Mines 2018
*/

#include <stdio.h>
#include <cmath>
#include <iostream>

//Set tolerance for the check
#define TOLERANCE 0.001

__global__ void spmv (int * ptr, int * indices, float * data, float * b, float * t) {
	int i = blockIdx.x;
	int tid = threadIdx.x;
	__shared__ float pSum[32];
	pSum[tid] = 0;

//utilize memory coalescing by using 32 threads and 32 data elements at a time
	for (int a = ptr[i] + tid; a<ptr[i+1]; a+= blockDim.x) {
		pSum[tid] +=  data[a] * b[indices[a]];
	}
	__syncthreads();//Sync threads for correctness

//utilize load balancing by using only 32 threads at a time and uses half the threads
	for (int b = blockDim.x/2; b > 0; b /=2) {
		if (tid < b) {
			pSum[tid] += pSum[tid+b];
		}
		__syncthreads();//Sync threads for correctness
	}
	t[i] = pSum[0];
}

main (int argc, char **argv) {
  FILE *fp;
  char line[1024];
  int *ptr, *indices;
  float *data, *b, *t;
  int i,j;
  int n; // number of nonzero elements in data
  int nr; // number of rows in matrix
  int nc; // number of columns in matrix

  // Open input file and read to end of comments
  if (argc !=2) abort();

  if ((fp = fopen(argv[1], "r")) == NULL) {
    abort();
  }

  fgets(line, 128, fp);
  while (line[0] == '%') {
    fgets(line, 128, fp);
  }

  // Read number of rows (nr), number of columns (nc) and
  // number of elements and allocate memory for ptr, indices, data, b and t.
  sscanf(line,"%d %d %d\n", &nr, &nc, &n);
  ptr = (int *) malloc ((nr+1)*sizeof(int));
  indices = (int *) malloc(n*sizeof(int));
  data = (float *) malloc(n*sizeof(float));
  b = (float *) malloc(nc*sizeof(float));
  t = (float *) malloc(nr*sizeof(float));

  // Read data in coordinate format and initialize sparse matrix
  int lastr=0;
  for (i=0; i<n; i++) {
    int r;
    fscanf(fp,"%d %d %f\n", &r, &(indices[i]), &(data[i]));
    indices[i]--;  // start numbering at 0
    if (r!=lastr) {
      ptr[r-1] = i;
      lastr = r;
    }
  }
  ptr[nr] = n;

  // initialize t to 0 and b with random data
  for (i=0; i<nr; i++) {
    t[i] = 0.0;
  }

  for (i=0; i<nc; i++) {
    b[i] = (float) rand()/1111111111;
  }

  // MAIN COMPUTATION, SEQUENTIAL VERSION
  for (i=0; i<nr; i++) {
    for (j = ptr[i]; j<ptr[i+1]; j++) {
      t[i] = t[i] + data[j] * b[indices[j]];
    }
  }

  // TODO: Compute result on GPU and compare output

//	A CUDA implementation of SpMV which optimizes for memory coalescing or load balancing.

//initialize and allocate memory for host copy of data
float * t_h;
t_h = (float *) malloc(nr*sizeof(float));

//initialize and allocate memory for device same set as host
int * ptr_d, * indices_d;
float * data_d, * b_d, *t_d;

hipMalloc((void**) & ptr_d, (nr+1)*sizeof(int));
hipMalloc((void**) & indices_d, n*sizeof(int));
hipMalloc((void**) & data_d, n*sizeof(float));
hipMalloc((void**) & b_d, nc*sizeof(float));
hipMalloc((void**) & t_d, nr*sizeof(float));

//copy data from host to device
hipMemcpy(ptr_d, ptr, (nr+1)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(indices_d, indices, n*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(data_d, data, n*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(b_d, b, nc*sizeof(float), hipMemcpyHostToDevice);
spmv<<<nr, 32>>>(ptr_d, indices_d, data_d, b_d, t_d);
hipMemcpy(t_h, t_d, nr*sizeof(float), hipMemcpyDeviceToHost);


//TODO: You should use the CPU implementation (sparse_matvec.c) to check whether the results produced by your GPU code are correct.

//Compares t (CPU) with t_h (GPU) to determine accuracy
int tfail = 0;
for (int i = 0; i < nr; i++) {
  if (abs(t_h[i] - t[i]) > TOLERANCE) {//take abs value and compare with tolerance
    tfail += 1;//if difference exceeds tolerance
  }
}

std::cout << "Number of failures: " << tfail <<"\n";//print the number of failures

}
