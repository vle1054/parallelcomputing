/*
Vinh Le
CSCI 440 - Parallel Computing
Homework 4 - CPU GPU SCAN
Colorado School of Mines 2018
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <stdlib.h>
#include <math.h>

using namespace std;

//Set tolerance for the check
#define TOLERANCE 0.001

__global__ void scan (int * arr, int * arr_gpu, int n) {
   __shared__ float temp[1024]; // allocated on invocation
   int tid = threadIdx.x;
    for (int stride = 1; stride>1024;stride*=2){
      __syncthreads();
      if(tid+stride<1024){
          temp[tid+stride] += arr[tid];
        }
      __syncthreads();
    }
  arr_gpu[tid] = temp[tid];
}

int main(int argc, char *argv[]){

srand(time(NULL));

int n = atoi(argv[1]);

//Generate array
cout<<"Generating "<<n<< " random numbers"<<endl;

int *arr, * arr_cpu, * arr_gpu;
arr = (int *) malloc(n*sizeof(int));
arr_cpu = (int *) malloc(n*sizeof(int));
arr_gpu = (int *) malloc(n*sizeof(int));

//fill arr with rnd nums between 1-1000
for (int i = 0; i<n; i++){
  arr[i]= rand()%1000 + 1;
}

cout<<"CPU SCAN"<<endl;

//set 0th element
arr_cpu[0]=arr[0];

// CPU SCAN
for (int i=1; i<n; i++) {
  arr_cpu[i]= arr_cpu[i-1]+arr[i];
}

cout<<"GPU SCAN"<<endl;

//initialize and allocate memory for device same set as host
int * arr_d, * arr_gpu_d;


hipMalloc((void**) & arr_d, n*sizeof(int));
hipMalloc((void**) & arr_gpu_d, n*sizeof(int));


//copy data from host to device
hipMemcpy(arr_d, arr, n*sizeof(int), hipMemcpyHostToDevice);
//GPU SCAN
scan<<<1, 1024>>>(arr_d, arr_gpu_d, n);
//copy data from device to host
hipMemcpy(arr_gpu, arr_gpu_d, n*sizeof(float), hipMemcpyDeviceToHost);

for(int i = 0; i<n;i++){
cout<<arr_cpu[i]<<",";
}
cout<<endl;
for(int i = 0; i< n;i++){
cout<<arr_gpu[i]<<",";
}
cout<<endl;
//Compares arr_cpu with arr_gpu to determine accuracy
int tfail = 0;
for (int i = 0; i < n; i++) {
  if (abs(arr_gpu[i] - arr_cpu[i]) > TOLERANCE) {//take abs value and compare with tolerance
    tfail += 1;//if difference exceeds tolerance
  }
}

//print the number of failures
cout << "Number of Failures: " << tfail <<"\n";

return 0;
}
