#include "hip/hip_runtime.h"
#include <stdio.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void countones(int *in, int *out) {

  __shared__ int *temp = 0;

  unsigned int tid = threadIdx.x;

  if (in[tid]==1){
    atomicadd(*temp,1);

  }

  __syncthreads();

  *out = *temp;
}

int main(int argc, char *argv[]){

  int *in, *out; // host copies in and cout
  int *d_in, *d_out; // device copies
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);
  // Alloc space for host copies of a, b, c and setup input values
  in = (int *)malloc(size);
  out = (int *)malloc(size);

  FILE *file = fopen(argv[1], "r");

  int data, row, col;
fscanf(file, "%d", &row)
fscanf(file, "%d", &col)


  for (int i = 0; i < row*col; i++)  {
    fscanf(file, "%d", &data);
    *in[i] = data;
  }

  fclose(file);

  int *in = *array;



  // Copy inputs to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  countones <<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>> (d_in, d_out);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}
