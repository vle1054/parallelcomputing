#include "hip/hip_runtime.h"
#include <string>
#include <fstream>
#include <iostream>
#include <Vector>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

__global__ void countones(int *in, int *out) {
  __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
  int gindex = threadIdx.x + blockIdx.x * blockDim.x;
  int lindex = threadIdx.x + radius;
  // Read input elements into shared memory
  temp[lindex] = in[gindex];
  if (threadIdx.x < RADIUS) {
    temp[lindex – RADIUS] = in[gindex – RADIUS];
    temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
  }
  // Synchronize (ensure all the data is available)
  __syncthreads();
  // Apply the stencil
  int result = 0;
  for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
  result += temp[lindex + offset];
  // Store the result
  out[gindex] = result;
}
int main(int argc, char *argv[]){
  int row, col, temp;
vector<int> array;
  string infile = argv[1];

  ifstream fin;
  fin.open(infile);
  fin >> row >> col;

for(ini=0;i<(row*col);i++){
fin<<temp;
array[i]=temp

  
}




  int *a, *b, *c; // host copies of a, b, c
  int *d_a, *d_b, *d_c; // device copies of a, b, c
  int size = N * sizeof(int);

  // Alloc space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
    // Alloc space for host copies of a, b, c and setup input values
  a = (int *)malloc(size); random_ints(a, N);
  b = (int *)malloc(size); random_ints(b, N);
  c = (int *)malloc(size);

  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  countones <<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>> (d_a, d_b, d_c);

  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

  // Cleanup
  free(a); free(b); free(c);
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
  return 0;
}
