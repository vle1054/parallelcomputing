
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void countones(int *in, int *out) {
  __shared__ int temp;
  unsigned int tid = threadIdx.x;
  if (in[tid]==1){
    atomicAdd(&temp,1);
  }
  __syncthreads();
  *out = temp;
}

int main(int argc, char *argv[]){

  FILE *file = fopen(argv[1], "r");
  int row, col;
  fscanf(file, "%d",&row);
  fscanf(file, "%d", &col);

  int size = row * col * sizeof(int);

  int *in, *out; // host copies in and cout
  in = (int *)malloc(size);
  out = (int *)malloc(sizeof(int));



    for (int i = 0; i < row*col; i++)  {
    fscanf(file, "%d", &in[i]);
  }

  fclose(file);

  int *d_in, *d_out; // device copies
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, sizeof(int));

  // Copy inputs to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  countones <<<1, row*col>>> (d_in, d_out);

  // Copy result back to host
  hipMemcpy(out, d_out, sizeof(int), hipMemcpyDeviceToHost);

  printf("There are %d ones.\n", *out);

  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}
