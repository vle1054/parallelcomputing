#include "hip/hip_runtime.h"
/*
Vinh Le
CSCI 440 - Parallel Computing
Homework 4 - CPU GPU SCAN
Colorado School of Mines 2018
*/
#include <iostream>
#include <ctime>
#include <stdlib.h>

using namespace std;

//Set tolerance for the check
#define TOLERANCE 0.001

__global__ void scan (int * arr, int * arr_gpu, int n) {



}

int main(int argc, char *argv[]){

srand(time(NULL));

int n = atoi(argv[1]);
int arr[n];
cout<<"Size of Array"<<sizeof(arr)<<endl;

//Generate array
cout<<"Generating "<<n<< " random numbers"<<endl;

int *arr, * arr_cpu, * arr_gpu;
arr = (int *) malloc(n*sizeof(int));
arr_cpu = (int *) malloc(n*sizeof(int));
arr_gpu = (int *) malloc(n*sizeof(int));
cout<<"Size of Array"<<sizeof(arr)<<endl;

//fill arr with rnd nums between 1-1000
for (int i = 0; i<n; i++){
  arr[i]= rand()%1000 + 1;
  cout<<arr[i]<<endl;
}
cout<<"Size of Array"<<sizeof(arr)<<endl;
cout<<"CPU SCAN"<<endl;

//set 0th element
arr_cpu[0]=arr[0];

// CPU SCAN
for (int i=1; i<n; i++) {
  arr_cpu[i]= arr_cpu[i-1]+arr[i];
}

cout<<"GPU SCAN"<<endl;


for(int i = 0; i<sizeof(arr_cpu)-1;i++){
cout<<arr_cpu[i]<<",";
}
cout<<endl;
for(int i = 0; i< sizeof(arr_gpu);i++){
cout<<arr_gpu[i]<<",";
}
cout<<endl;
//Compares arr_cpu with arr_gpu to determine accuracy
int tfail = 0;
for (int i = 0; i < n; i++) {
  if (abs(arr_gpu[i] - arr_cpu[i]) > TOLERANCE) {//take abs value and compare with tolerance
    tfail += 1;//if difference exceeds tolerance
  }
}

//print the number of failures
cout << "Number of Failures: " << tfail <<"\n";

return 0;
}
