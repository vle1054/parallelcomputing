#include "hip/hip_runtime.h"
/*
Vinh Le
CSCI 440 - Parallel Computing
Homework 4 - CPU GPU SCAN
Colorado School of Mines 2018
*/


#include <iostream>
#include <ctime>
#include <stdlib.h>

using namespace std;

//Set tolerance for the check
#define TOLERANCE 0.001

//avoid bank conflicts
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \
 ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

__global__ void SCAN (int * arr, int * arr_gpu, int * n) {
  extern __shared__ float temp[];// allocated on invocation
  int thid = threadIdx.x;
  int offset = 1;

  int ai = thid;
  int bi = thid + (n/2);
  int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
  int bankOffsetB = CONFLICT_FREE_OFFSET(ai);
  temp[ai + bankOffsetA] = arr[ai];
  temp[bi + bankOffsetB] = arr[bi];

  int *d;
  for (d = n>>1; d > 0; d >>= 1){ // build sum in place up the tree

  __syncthreads();

  if (thid < d){
    int ai = offset*(2*thid+1)-1;
    int bi = offset*(2*thid+2)-1;
    ai += CONFLICT_FREE_OFFSET(ai);
    bi += CONFLICT_FREE_OFFSET(bi);

    temp[bi] += temp[ai];
  }

  offset *= 2;

  }
  if (thid==0) { temp[n – 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0; }
  for (int d = 1; d < n; d *= 2){ // traverse down tree & build scan

     offset >>= 1;
     __syncthreads();
     if (thid < d){
       int ai = offset*(2*thid+1)-1;
       int bi = offset*(2*thid+2)-1;
       ai += CONFLICT_FREE_OFFSET(ai);
       bi += CONFLICT_FREE_OFFSET(bi);

       float t = temp[ai];
       temp[ai] = temp[bi];
       temp[bi] += t;
      }
   }
   __syncthreads();

  arr_gpu[ai] = temp[ai + bankOffsetA];
  arr_gpu[bi] = temp[bi + bankOffsetB];

}

int main(int argc, char *argv[]){

srand(time(NULL));

int * n;
n = (int *) malloc(sizeof(int));
n = atoi(argv[1]);

//Generate array
cout<<"Generating "<<n<< " random numbers"<<endl;

int * arr, * arr_cpu, * arr_gpu;
arr = (int *) malloc(n*sizeof(int));
arr_cpu = (int *) malloc(n*sizeof(int));
arr_gpu = (int *) malloc(n*sizeof(int));

//fill arr with rnd nums between 1-1000
for (int i = 0; i<n; i++){
  arr[i]= rand()%1000 + 1;
}

cout<<"CPU SCAN"<<endl;

//set 0th element
arr_cpu[0]=arr[0];

// CPU SCAN
for (int i=1; i<n; i++) {
  arr_cpu[i]= arr_cpu[i-1]+arr[i];
}

//initialize and allocate memory for device same set as host
int * arr_d, * arr_gpu_d;
int * n_d;

hipMalloc((void**) & arr_d, n*sizeof(int));
hipMalloc((void**) & arr_gpu_d, n*sizeof(int));
hipMalloc((void**) & n_d, sizeof(int));

cout<<"GPU SCAN"<<endl;

//copy data from host to device
hipMemcpy(arr_d, arr, n*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(n_d, n, sizeof(int), hipMemcpyHostToDevice);
//GPU SCAN
scan<<<n, 32>>>(arr_d, arr_gpu_d, n_d);
//copy data from device to host
hipMemcpy(arr_gpu, arr_gpu_d, n*sizeof(float), hipMemcpyDeviceToHost);


//Compares arr_cpu with arr_gpu to determine accuracy
int tfail = 0;
for (int i = 0; i < n; i++) {
  if (abs(arr_gpu[i] - arr_cpu[i]) > TOLERANCE) {//take abs value and compare with tolerance
    tfail += 1;//if difference exceeds tolerance
  }
}

//print the number of failures
cout << "Number of Failures: " << tfail <<"\n";

return 0;
}
