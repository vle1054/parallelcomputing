/*
Vinh Le
CSCI 440 - Parallel Computing
Homework 2.2 - transpose matrix
Colorado School of Mines 2018
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void transpose(int *in, int *out, int row, int col) {
  unsigned int tid = threadIdx.x;
  if( tid<(row*col)){
    int newid = ((tid%row)*col+(tid/row));
    out[newid] = in[tid];
  }
  __syncthreads();


}

int main(int argc, char *argv[]){

  FILE *file = fopen(argv[1], "r");
  int row, col;
  fscanf(file, "%d",&row);
  fscanf(file, "%d", &col);

  int size = row * col * sizeof(int);

  int *in, *out; // host copies in and cout
  in = (int *)malloc(size);
  out = (int *)malloc(size);

  for (int i = 0; i < row*col; i++)  {
    fscanf(file, "%d", &in[i]);
  }

  fclose(file);

  int *d_in, *d_out; // device copies
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  // Copy inputs to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

  // Launch add() kernel on GPU
  transpose <<<1, row*col>>> (d_in, d_out, row, col);

  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);


  for (int i = 0 ; i < row*col;i++){
    if (i%row==0){
      printf("\n");
    }
    printf("%d ", in[i]);

  }
  printf("\n");

  for (int i = 0 ; i < row*col;i++){
    if (i%col==0){
      printf("\n");
    }

    printf("%d ", out[i]);
  }


  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}
