/*
	Vinh Le
	CSCI 440 - Parallel Computing
	Homework 4 - CPU GPU SCAN
	Colorado School of Mines 2018
*/

#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include <stdlib.h>
#include <math.h>
#include <cstdio>

using namespace std;

//Set tolerance for the check
#define TOLERANCE 0.001
#define BLOCK_SIZE 1024


__global__ void scan (int * arr, int * arr_gpu, int * aux, int n){
	
	__shared__ float temp[BLOCK_SIZE];
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	
	if (i < n && i > 0) {
		temp[tid] = arr[i-1];
		}else{
		temp[0]= 0;
	}
	int tempint;
	
	for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
		__syncthreads();
		if(tid>=stride){
			tempint = temp[tid - stride];
		}
		__syncthreads();
		if(tid>=stride){
			temp[tid] += tempint;
		}
	}
	__syncthreads();
	
	if(i < n) {
		arr_gpu[i] = temp[tid];
	}
	if(tid == 0 && aux != NULL){
		aux[bid]=temp[1023];
	}
}


__global__ void finish (int * arr,int *aux, int NUM_BLOCK){
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	if (bid>=1){
		arr[bid*BLOCK_SIZE+tid] += aux[bid];
	}
	__syncthreads();
}


/*
	__global__ void finish (int * arr, int NUM_BLOCK){
	int tid = threadIdx.x;
	for(int j = 1; j<NUM_BLOCK;j++){
	arr[j*BLOCK_SIZE+tid] += arr[j*BLOCK_SIZE-1];
	__syncthreads();
	}
	}
*/
int main(int argc, char *argv[]){
	
	srand(time(NULL));
	
	int n = atoi(argv[1]);
	
	//Generate array
	cout<<"Generating "<<n<< " random numbers"<<endl;
	
	int *arr, * arr_cpu, * arr_gpu;
	arr = (int *) malloc(n*sizeof(int));
	arr_cpu = (int *) malloc(n*sizeof(int));
	arr_gpu = (int *) malloc(n*sizeof(int));
	
	//fill arr with rnd nums between 1-1000
	for (int i = 0; i<n; i++){
		arr[i]= rand()%1000 + 1;
		//arr[i]=1;//for debug
	}
	
	cout<<"CPU SCAN"<<endl;
	
	//set 0th element
	arr_cpu[0]=0;
	
	// CPU SCAN
	for (int i=1; i<n; i++) {
		arr_cpu[i]= arr_cpu[i-1]+arr[i-1];
	}
	
	cout<<"GPU SCAN"<<endl;
	
	//initialize and allocate memory for device same set as host
	int * arr_d, * arr_gpu_d;
	
	hipMalloc((void**) & arr_d, n*sizeof(int));
	hipMalloc((void**) & arr_gpu_d, n*sizeof(int));
	
	int NUM_BLOCK = ceil((float)n/BLOCK_SIZE);
	
	int * aux_d;
	hipMalloc((void**) & aux_d, NUM_BLOCK*sizeof(int));
	
	//copy data from host to device
	hipMemcpy(arr_d, arr, n*sizeof(int), hipMemcpyHostToDevice);
	
	//GPU SCAN
	scan<<<NUM_BLOCK, BLOCK_SIZE>>>(arr_d, arr_gpu_d, aux_d, n);//Scan main array
	scan<<<1, BLOCK_SIZE>>>(aux_d, aux_d, NULL, n);//scan aux array
	finish<<<NUM_BLOCK, BLOCK_SIZE>>>(arr_gpu_d, aux_d, NUM_BLOCK);//add aux array to main array
	
	//copy data from device to host
	hipMemcpy(arr_gpu, arr_gpu_d, n*sizeof(int), hipMemcpyDeviceToHost);
	
	//Compares arr_cpu with arr_gpu to determine accuracy
	int tfail = 0;
	for (int i = 0; i < n; i++) {
		if (abs(arr_gpu[i] - arr_cpu[i]) > TOLERANCE) {//take abs value and compare with tolerance
			tfail += 1;//if difference exceeds tolerance
		}
	}
	
	//print the number of failures
	cout << "Number of Failures: " << tfail <<"\n";
	
	return 0;
}
