/*
Vinh Le
CSCI 440 - Parallel Computing
Colorado School of Mines 2018
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void transpose(int *in, int *out, int row, int col) {
  __shared__ int *temp;
  unsigned int tid = threadIdx.x;
if( tid<(row*col)){
 temp[(tid/row)*row+(tid%row)] = in[tid];

printf("tid %d was transposed to %d", tid, ((tid/row)*row+(tid%row)));
  }
__syncthreads();
  *out = *temp;
}

int main(int argc, char *argv[]){

  FILE *file = fopen(argv[1], "r");
  int row, col;
  fscanf(file, "%d",&row);
  fscanf(file, "%d", &col);

  int size = row * col * sizeof(int);

  int *in, *out; // host copies in and cout
  in = (int *)malloc(size);
  out = (int *)malloc(size);

  for (int i = 0; i < row*col; i++)  {
    fscanf(file, "%d", &in[i]);
  }

  fclose(file);

  int *d_in, *d_out; // device copies
  hipMalloc((void **)&d_in, size);
  hipMalloc((void **)&d_out, size);

  // Copy inputs to device
  hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);

printf("running transpose\n");
  // Launch add() kernel on GPU
  transpose <<<1, row*col>>> (d_in, d_out, row, col);

printf("Finish transpose\n");
  // Copy result back to host
  hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);

  for (int i = 0; i< col;i++){
    for (int j=0; j< row;j++){
      printf("%d",in[i*col+j]);
    }
    printf("\n");
  }


  for (int i = 0; i< row;i++){
    for (int j=0; j< col;j++){
      printf("%d",out[i*row+j]);
    }
    printf("\n");
  }
  // Cleanup
  free(in); free(out);
  hipFree(d_in); hipFree(d_out);
  return 0;
}
