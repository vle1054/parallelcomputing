#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <iostream>

#define TOLERANCE 0.001

__global__ void spmv (int * ptr, int * indices, float * data, float * b, float * t) {
	int i = blockIdx.x;
	int tid = threadIdx.x;
	__shared__ float pSum[32];
	pSum[tid] = 0;
	for (int j = ptr[i] + tid; j<ptr[i+1]; j+= blockDim.x) {
		pSum[tid] +=  data[j] * b[indices[j]];
	}
	__syncthreads();
	for (int z = blockDim.x/2; z > 0; z /=2) {
		if (tid < z) {
			pSum[tid] += pSum[tid+z];
		}
		__syncthreads();
	}
	t[i] = pSum[0];
}

main (int argc, char **argv) {
  FILE *fp;
  char line[1024];
  int *ptr, *indices;
  float *data, *b, *t;
  int i,j;
  int n; // number of nonzero elements in data
  int nr; // number of rows in matrix
  int nc; // number of columns in matrix

  // Open input file and read to end of comments
  if (argc !=2) abort();

  if ((fp = fopen(argv[1], "r")) == NULL) {
    abort();
  }

  fgets(line, 128, fp);
  while (line[0] == '%') {
    fgets(line, 128, fp);
  }

  // Read number of rows (nr), number of columns (nc) and
  // number of elements and allocate memory for ptr, indices, data, b and t.
  sscanf(line,"%d %d %d\n", &nr, &nc, &n);
  ptr = (int *) malloc ((nr+1)*sizeof(int));
  indices = (int *) malloc(n*sizeof(int));
  data = (float *) malloc(n*sizeof(float));
  b = (float *) malloc(nc*sizeof(float));
  t = (float *) malloc(nr*sizeof(float));
  //add malloc for host t
  float * t_h;
  t_h = (float *) malloc(nr*sizeof(float));


  // Read data in coordinate format and initialize sparse matrix
  int lastr=0;
  for (i=0; i<n; i++) {
    int r;
    fscanf(fp,"%d %d %f\n", &r, &(indices[i]), &(data[i]));
    indices[i]--;  // start numbering at 0
    if (r!=lastr) {
      ptr[r-1] = i;
      lastr = r;
    }
  }
  ptr[nr] = n;

  // initialize t to 0 and b with random data
  for (i=0; i<nr; i++) {
    t[i] = 0.0;
  }

  for (i=0; i<nc; i++) {
    b[i] = (float) rand()/1111111111;
  }

  // MAIN COMPUTATION, SEQUENTIAL VERSION
  for (i=0; i<nr; i++) {
    for (j = ptr[i]; j<ptr[i+1]; j++) {
      t[i] = t[i] + data[j] * b[indices[j]];
    }
  }

  // TODO: Compute result on GPU and compare output

int * ptr_d, * indices_d;
float * data_d, * b_d, *t_d;

hipMalloc((void**) & ptr_d, (nr+1)*sizeof(int));
hipMalloc((void**) & indices_d, n*sizeof(int));
hipMalloc((void**) & data_d, n*sizeof(float));
hipMalloc((void**) & b_d, nc*sizeof(float));
hipMalloc((void**) & t_d, nr*sizeof(float));

hipMemcpy(ptr_d, ptr, (nr+1)*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(indices_d, indices, n*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(data_d, data, n*sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(b_d, b, nc*sizeof(float), hipMemcpyHostToDevice);
spmv<<<nr, 32>>>(ptr_d, indices_d, data_d, b_d, t_d);
hipMemcpy(t_h, t_d, nr*sizeof(float), hipMemcpyDeviceToHost);




int fail = 0;
for (int k = 0; k < nr; k++) {
  if (abs(t_h[k] - t[k]) > TOLERANCE) {
    fail += 1;
  }
}

std::cout << "Number of failures: " <<fail <<"\n";

}
